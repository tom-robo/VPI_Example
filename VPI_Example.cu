#include <iostream> 
#include <vector>
#include <sstream>
#include <iomanip>

#include <hip/hip_runtime.h> 

#include <vpi/VPI.h>
#include <vpi/algo/OpticalFlowPyrLK.h>
#include <vpi/algo/GaussianPyramid.h>
#include <vpi/algo/HarrisCorners.h>
#include <vpi/OpenCVInterop.hpp>
#include <vpi/algo/HarrisCorners.h>
#include <vpi/algo/ConvertImageFormat.h>
#include <vpi/Image.h>

#include <opencv2/core/mat.hpp>

#include <opencv2/imgcodecs.hpp>
#include <opencv2/imgproc.hpp> 

inline void checkVPIError(VPIStatus stmt, const char *file, int line)
{
    VPIStatus status__ = (stmt);                            
    if (status__ != VPI_SUCCESS)                            
    {                                                       
        char buffer[VPI_MAX_STATUS_MESSAGE_LENGTH];         
        vpiGetLastStatusMessage(buffer, sizeof(buffer));    
        std::ostringstream ss;                              
        ss << vpiStatusGetName(status__) << ": " << buffer 
                << " file: " << file << ":" << line << "\n"; 
        throw std::runtime_error(ss.str());                 
    }                                                       
}

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        std::stringstream err; 
        err << "ERROR: GPUassert: " << hipGetErrorString(code) 
                << " " << file << ":" << line << std::endl;
        throw std::runtime_error(err.str());
    }
}

#define gpuErrchk(ans) { \
            gpuAssert((ans), __FILE__, __LINE__); \
        }

#define CHECK_VPI(STMT) do  {  \
        STMT; \
        checkVPIError(STMT, __FILE__, __LINE__); \
    } while (0);


class tracker
{
public: 

    tracker(int w, int h, float s, int l, int max_corners) : 
        width(w), 
        height(h), 
        scale(s), 
        levels(l), 
        maxCorners(max_corners)
    {
        currMat = cv::Mat(height, width,  CV_8UC3); 
        grayMat = cv::Mat(height, width,  CV_8UC1); 
        equalHist = cv::Mat(height, width,  CV_8UC1); 
        harrisMat = cv::Mat(height, width, CV_8UC3); 

        CHECK_VPI(vpiImageCreate(width, height, format, 0,&prevImage)); 
        CHECK_VPI(vpiImageCreate(width, height, format, 0,&inputImage));
        CHECK_VPI(vpiImageCreateWrapperOpenCVMat(currMat, 0, &currImage));

        CHECK_VPI(vpiImageCreateWrapperOpenCVMat(harrisMat, 0, &wrappedHarris));
        CHECK_VPI(vpiImageCreate(width, height, VPI_IMAGE_FORMAT_S16, 0, &inputHarris));

        CHECK_VPI(vpiPyramidCreate(width, height, format, levels, scale, 0, &pyrPrevFrame));
        CHECK_VPI(vpiPyramidCreate(width, height, format, levels, scale, 0, &pyrCurFrame));

        CHECK_VPI(vpiArrayCreate(maxCorners, VPI_ARRAY_TYPE_KEYPOINT_F32, 0, &prevFeatures));
        CHECK_VPI(vpiArrayCreate(maxCorners, VPI_ARRAY_TYPE_KEYPOINT_F32, 0, &currFeatures));
        CHECK_VPI(vpiArrayCreate(maxCorners, VPI_ARRAY_TYPE_U32, 0, &scores));
    
        CHECK_VPI(vpiCreateOpticalFlowPyrLK(backend,
                                        width, height,
                                        format,
                                        levels,scale, 
                                        &optflow));

        CHECK_VPI(vpiInitOpticalFlowPyrLKParams(&lkParams));

        CHECK_VPI(vpiCreateHarrisCornerDetector(backend, width, height, &harris));
        CHECK_VPI(vpiInitHarrisCornerDetectorParams(&harrisParams));

        CHECK_VPI(vpiStreamCreate(0, &stream));

    }

    ~tracker()
    {

    }

    void createStatusArray()
    {
        // CHECK_VPI(vpiArrayCreate(maxCorners, VPI_ARRAY_TYPE_U8, 0, &featStatus));
        
        featStatusData.bufferType = VPI_ARRAY_BUFFER_CUDA_AOS;
        featStatusData.buffer.aos.sizePointer = &numStatusPoints; 
        featStatusData.buffer.aos.capacity = maxCorners; 
        featStatusData.buffer.aos.strideBytes = maxCorners; 

        gpuErrchk(hipMalloc(&featStatusData.buffer.aos.data, maxCorners)); 

        featStatusData.buffer.aos.type = VPI_ARRAY_TYPE_U8;

        CHECK_VPI(vpiArrayCreateWrapper(&featStatusData, VPI_BACKEND_CUDA, &featStatus)); 
        
        std::cout << "Created array wrapper" << std::endl;;
    }


    void destroyStatusArray()
    {
        if (featStatus != NULL) 
        {
            vpiArrayDestroy(featStatus);
            featStatus = NULL; 
        }

        gpuErrchk(hipFree(featStatusData.buffer.aos.data));
    }

    void makeCurrGaussPyramid()
    {
        // Adding image to input for LKTracker
        CHECK_VPI(vpiImageSetWrappedOpenCVMat(currImage, currMat));
        CHECK_VPI(vpiSubmitConvertImageFormat(stream, backend, currImage, inputImage, NULL));

        // Making pyrCurFrame gaussian pyramid; 
        CHECK_VPI(vpiSubmitGaussianPyramidGenerator(stream, backend, inputImage, pyrCurFrame, VPI_BORDER_CLAMP));
    }

// Convert colour image to grayscale and add to vpi
    void setInputImage(std::string filename)
    {
        cv::Mat image; 

        try
        {
            image = cv::imread(filename);

        }
        catch (const cv::Exception& e)
        {
            std::cerr << "ERROR: Cannot load image: " << filename << " Error: " << e.what() << std::endl; 
        }

//        std::cout << "INFO: Loaded image: " << filename << " Width: " << image.cols << " Height: " << image.rows << std::endl;

        try
        {
            cv::cvtColor(image, grayMat, cv::COLOR_RGB2GRAY);
        }
        catch (cv::Exception& e)
        {
            std::cerr << "ERROR: Cannot convert colour input image to grayscale. Error: " << e.what() << std::endl;
        }

        try
        {
            cv::equalizeHist(grayMat, equalHist);
            cv::cvtColor(equalHist, currMat, cv::COLOR_GRAY2RGB);
        }
        catch (cv::Exception& e)
        {
            std::cerr << "ERROR: Canot convert grayscale mat to RGB (3-channels). Errror: " << e.what() << std::endl; 
        }

        harrisMat = currMat.clone(); 
    }

    int trackPoints(bool zeroStatusBuffer)
    {
        numTrackedPoints=0; 

        makeCurrGaussPyramid(); 

        if (!first)
        {

            // debugCheckStatusHasBeenSetToZero(numFeatures2Track); 

            CHECK_VPI(vpiSubmitOpticalFlowPyrLK(stream, 0, optflow, pyrPrevFrame, pyrCurFrame, 
                                                    prevFeatures, currFeatures, featStatus, &lkParams));

            CHECK_VPI(vpiStreamSync(stream));
            setOutputPoints(zeroStatusBuffer);
        
        }
        else
        { 
            first = false; 
            return 0; 
        }

        computePointsToTrack();
        swapPyramids(); 

        return numTrackedPoints; 
    }

    int getNumTrackedPoints()
    {
        return numTrackedPoints; 
    }

    int getNumFeatures2Track()
    {
        return numFeatures2Track;
    }

private: 

    void debugCheckStatusHasBeenSetToZero(int num_points)
    {
        VPIArrayData currStatusBuff; 

        int zeroedElements = 0;
        int nonZeroedElements = 0;  

        CHECK_VPI(vpiArrayLockData(featStatus, VPI_LOCK_READ, VPI_ARRAY_BUFFER_HOST_AOS, &currStatusBuff));

        unsigned char* currStatusArray = (unsigned char*)currStatusBuff.buffer.aos.data;

        for (int i = 0; i<num_points; i++)
        {
            unsigned char status = currStatusArray[i];

            if (status == 0)
            {
                zeroedElements++;
            }
            else
            {
                nonZeroedElements++;
            }
        }

        CHECK_VPI(vpiArrayUnlock(featStatus));

        if (zeroedElements == num_points)
        {
            std::cout << "SUCCESS: Point to be checked: " << num_points 
                        << "number of non-zero points: " << nonZeroedElements
                        << " number of zeroed elements: " << zeroedElements << std::endl; 

        }
        else
        {
            std::cout << "FAILURE: Point to be checked: " << num_points 
                        << "number of non-zero points: " << nonZeroedElements
                        << " number of zeroed elements: " << zeroedElements
                        << " Not all points have been zeroed! " << std::endl; 
        }
    }

    void setOutputPoints(bool zeroStatusBuffer)
    {
        
        VPIArrayData currFeaturesBuff; 
        VPIArrayData prevFeaturesBuff; 
        VPIArrayData currStatusBuff; 

        CHECK_VPI(vpiArrayLockData(currFeatures, VPI_LOCK_READ, VPI_ARRAY_BUFFER_HOST_AOS, &currFeaturesBuff));
        CHECK_VPI(vpiArrayLockData(prevFeatures, VPI_LOCK_READ, VPI_ARRAY_BUFFER_HOST_AOS, &prevFeaturesBuff));
        CHECK_VPI(vpiArrayLockData(featStatus, VPI_LOCK_READ_WRITE, VPI_ARRAY_BUFFER_HOST_AOS, &currStatusBuff));

        VPIKeypointF32* currKeypointsArray = (VPIKeypointF32 *)currFeaturesBuff.buffer.aos.data;
        VPIKeypointF32* prevKeypointsArray = (VPIKeypointF32 *)prevFeaturesBuff.buffer.aos.data;

        unsigned char* currStatusArray = (unsigned char*)currStatusBuff.buffer.aos.data;

        currTrackedPoints.clear(); 
        prevTrackedPoints.clear(); 

        for (int i = 0; i<numFeatures2Track; i++)
        {
            unsigned char status = currStatusArray[i];
            VPIKeypointF32 currPoint = currKeypointsArray[i]; 
            VPIKeypointF32 prevPoint = prevKeypointsArray[i]; 

            if (status == 0)
            {
                currTrackedPoints.push_back({prevPoint.x, prevPoint.y}); 
                prevTrackedPoints.push_back({prevPoint.x, prevPoint.y});

                numTrackedPoints++;
            }

            if (zeroStatusBuffer)
            {
                currStatusArray[i] = 0; 
            }
        }

/// Setting statusPtr to zero via hipMemset; 
        gpuErrchk(hipMemset(statusPtr, 0, maxCorners)); 

        CHECK_VPI(vpiArrayUnlock(currFeatures));
        CHECK_VPI(vpiArrayUnlock(prevFeatures));
        CHECK_VPI(vpiArrayUnlock(featStatus));
    }

    void swapPyramids()
    {
        std::swap(pyrPrevFrame, pyrCurFrame);
    }

    void computePointsToTrack()
    {
        // Adding image to input for Harris Corners (needs to be S16 format unlike LKTracker)
        CHECK_VPI(vpiImageSetWrappedOpenCVMat(wrappedHarris, harrisMat));
        CHECK_VPI(vpiSubmitConvertImageFormat(stream, VPI_BACKEND_CUDA, 
                                                    wrappedHarris, inputHarris, NULL));

        CHECK_VPI(vpiArraySetSize(prevFeatures, maxCorners)); 

        CHECK_VPI(vpiSubmitHarrisCornerDetector(stream, backend, harris, inputHarris,
                                                prevFeatures, scores, &harrisParams));
        CHECK_VPI(vpiStreamSync(stream));

        VPIArrayData prevPointsBuff; 
        CHECK_VPI(vpiArrayLockData(prevFeatures, VPI_LOCK_READ, VPI_ARRAY_BUFFER_HOST_AOS, &prevPointsBuff));
        numFeatures2Track = *prevPointsBuff.buffer.aos.sizePointer;
        
        numStatusPoints = numFeatures2Track; 

        CHECK_VPI(vpiArrayUnlock(prevFeatures));
//        std::cout << " Number of detected Points: " << numFeatures2Track << std::endl;

    }


private: 
    int width; 
    int height; 
    float scale; 
    int levels; 
    int maxCorners; 

    bool first = true; 

    int32_t numFeatures2Track=0;
    int32_t numTrackedPoints=0;

    std::vector<cv::Point2f> prevTrackedPoints; 
    std::vector<cv::Point2f> currTrackedPoints; 

    VPIStream stream;

    cv::Mat currMat; 
    cv::Mat grayMat; 
    cv::Mat equalHist; 

    VPIImage inputHarris; 
    VPIImage wrappedHarris; 
    cv::Mat harrisMat; 

    VPIImage currImage; 
    VPIImage inputImage; 
    VPIImage prevImage;

    VPIBackend backend = VPI_BACKEND_CUDA; 
    VPIImageFormat format = VPI_IMAGE_FORMAT_U8;

    unsigned char* statusPtr; 
    int32_t numStatusPoints=0;

    VPIPyramid pyrPrevFrame=NULL;
    VPIPyramid pyrCurFrame=NULL; 

    VPIArray prevFeatures=NULL;
    VPIArray currFeatures=NULL;
    VPIArray scores=NULL;

    VPIArray featStatus=NULL;
    VPIArrayData featStatusData; 

    VPIOpticalFlowPyrLKParams lkParams;
    VPIPayload optflow=NULL;

    VPIPayload harris=NULL;
    VPIHarrisCornerDetectorParams harrisParams;

}; 

void reuseStatusArray(bool zeroStatusBuffer, int width, int height, float scale, int levels, int max_corners, int n)
{
    std::cout << "_____REUSING STATUS ARRAY. "
                << " ZEROING STATUS BUFFER: " << zeroStatusBuffer << "_____" << std::endl;

    tracker track(width, height, scale, levels, max_corners); 

    track.createStatusArray();

    for (int i=1; i<n; ++i)
    {
        std::stringstream filename; 
        filename << "./../dashcam-" << std::setw(3) << std::setfill('0') << i << ".jpg"; 

        std::cout << " Into tracking loop"; 

        track.setInputImage(filename.str());

        std::cout << " Set input image"; 

        int numPointsTracked = track.trackPoints(zeroStatusBuffer);

        std::cout << " Tracked points"; 

        std::cout << i << ": Point in prev array: " << track.getNumFeatures2Track() << " number of points tracked: " << track.getNumTrackedPoints() << std::endl; 
    }

    track.destroyStatusArray(); 
}

void createDestroyStatusArray(bool zeroStatusBuffer, int width, int height, float scale, int levels, int max_corners, int n)
{
    std::cout << "____CREATING/ DESTROYING STATUS ARRAY EVERY O.F. CYCLE. "
                << " ZEROING STATUS BUFFER: " << zeroStatusBuffer << "_____" << std::endl;

    tracker track(width, height, scale, levels, max_corners); 

    for (int i=1; i<n; i++)
    {
        std::stringstream filename; 
        filename << "./../dashcam-" << std::setw(3) << std::setfill('0') << i << ".jpg"; 

        track.setInputImage(filename.str()); 

        track.createStatusArray();

        int numPointsTracked = track.trackPoints(zeroStatusBuffer);
    
        track.destroyStatusArray(); 

        std::cout << i << ": Point in prev array: " << track.getNumFeatures2Track() << " number of points tracked: " << track.getNumTrackedPoints() << std::endl; 

    }
}

int main(int argc, char **argv) 
{
    int width = 1280; 
    int height = 720;
    float scale = 0.5; 
    int maxCorners = 8192;
    int levels = 4; 

    int n = 20; 

    reuseStatusArray(true, width, height, scale, levels, maxCorners, n); 
    createDestroyStatusArray(true, width, height, scale, levels, maxCorners, n);

    reuseStatusArray(false, width, height, scale, levels, maxCorners, n);
    createDestroyStatusArray(false, width, height, scale, levels, maxCorners, n);
 
}
